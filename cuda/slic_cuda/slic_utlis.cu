#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hiprand/hiprand_kernel.h>
#include "slic_utils.cuh"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/transform.h>
#include <thrust/functional.h>

__global__ void find_coords_opt(
    const int64_t * __restrict__ segment,
    const int64_t * __restrict__ segment_unique,
    const int64_t * __restrict__ segment_count,
    const int * __restrict__ pos,
    int H,
    int W,
    int num_segment,
    int64_t * pixel_coord,
    int64_t * segment_index
){
    // every block is responsible for origin data
    // every thread is responsible for a pixel
    // every thread will check if the pixel belongs to the segment
    // if yes, then write the pixel to the pixel_coord
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= H*W) return;

    // load block shared memory


    int i = idx / W;
    int j = idx % W;

}


__global__ void find_coords(
    const int64_t * __restrict__ segment,
    const int64_t * __restrict__ segment_unique,
    const int64_t * __restrict__ segment_count,
    const int * __restrict__ pos,
    int H,
    int W,
    int num_segment,
    int64_t * pixel_coord,
    int64_t * segment_index
){
    int segment_idx = blockIdx.x;
    if (segment_idx >= num_segment) return;

    // a shared memory to store the start index of each segment
    __shared__ int index;
    if(threadIdx.x == 0)
    {
        index = pos[segment_idx];
    }
    __syncthreads();

    // int start = pos[segment_idx];
    int count = segment_count[segment_idx];
    int segment_id = segment_unique[segment_idx];

    for(int pixel_idx = threadIdx.x; pixel_idx < H*W; pixel_idx += blockDim.x){
        int i = pixel_idx / W;
        int j = pixel_idx % W;
        // printf("segment_idx: %d, pixel_idx: %d\n", segment_id, pixel_idx);
        if (segment[pixel_idx] == segment_id){
            int idx = atomicAdd(&index, 1);
            // printf("idx: %d\n", index);
            pixel_coord[idx*2] = i;
            pixel_coord[idx*2 + 1] = j;
            segment_index[idx] = segment_id;
            // atomicAdd(&index, 1);
        }
    }
}

void generate_sample(torch::Tensor segment,
                    torch::Tensor segment_unique,
                    torch::Tensor segment_count,
                    torch::Tensor pixel_coords,
                    torch::Tensor segment_index,
                    // at::Tensor
                    int H,
                    int W,
                    int num_sample,
                    float sample_ratio){

    CHECK_INPUT(segment);
    CHECK_INPUT(segment_unique);
    CHECK_INPUT(segment_count);
    CHECK_INPUT(pixel_coords);
    CHECK_INPUT(segment_index);

    int num_segment = segment_unique.size(0);
    int num_point = segment.size(0);

    int sample_count = 0;
    thrust::device_vector<int64_t> segment_count_thrust(segment_count.data_ptr<int64_t>(), segment_count.data_ptr<int64_t>() + num_segment);
    // sample_count = thrust::reduce(segment_count_thrust.begin(), segment_count_thrust.end());

    int* pos;
    hipMallocManaged(&pos, num_segment * sizeof(int));

    // compute results start position
    for (int i=0; i<num_segment; ++i)
    {
        pos[i] = thrust::reduce(segment_count_thrust.begin(), segment_count_thrust.begin() + i);
    }
    hipDeviceSynchronize();

    // printf("befor kernel function : %s\n",hipGetErrorString(hipGetLastError()));

    int num_thread = 512;
    int num_blocks = num_segment;
    // printf("number of blocks: %d\n", num_blocks);
    find_coords<<<num_blocks, num_thread>>>(segment.data_ptr<int64_t>(), 
                                                    segment_unique.data_ptr<int64_t>(), 
                                                    segment_count.data_ptr<int64_t>(), 
                                                    pos,
                                                    H, W, num_segment, pixel_coords.data_ptr<int64_t>(), segment_index.data_ptr<int64_t>());
    hipDeviceSynchronize();
    // printf("after kernel function : %s\n",hipGetErrorString(hipGetLastError()));
    
    hipFree(pos);
}