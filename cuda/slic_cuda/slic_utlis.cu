#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hiprand/hiprand_kernel.h>
#include "slic_utils.cuh"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/transform.h>
#include <thrust/functional.h>

// generate random int, no repeat
__global__ void generate_random_int(int min, int max, int N, int* results)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N)
    {
        hiprandState_t state;
        hiprand_init(0, idx, 0, &state);

        // Generate unique random intergers in the range [min, max]
        int range = max - min;
        int r = hiprand(&state) % range + min;

        results[idx] = r;

    }
}

__global__ void countIndexKernel(
    const int * __restrict__ d_tensor,
    int rows, int cols,  int d_int, 
    int* count){
    // 
    // int i = blockIdx.x * blockDim.x + threadIdx.x;
    // int j = blockIdx.y * blockDim.y + threadIdx.y;

    // // output shape is unknow
    // if (i >= rows || j >= cols) return;
    int ii= 0;
    int global_idx = blockIdx.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;
    int i = global_idx / cols;
    int j = global_idx % cols;
    // printf("i: %d, j: %d\n", i, j);

    if (i >= rows || j >= cols) return;
    // printf("num idx %d \n ", i*cols + j);
    __syncthreads();

    if (d_tensor[i * cols + j] == d_int) {
        // atomicAdd(&count, 1);
        // *count += 1;
        atomicAdd(count, 1);
        // ii+=1;
    }
}

__global__ void findIndexKernel(
    const int * __restrict__ d_tensor,
    int rows, int cols, int d_int, int* result) {

    // int i = blockIdx.x * blockDim.x + threadIdx.x;
    // int j = blockIdx.y * blockDim.y + threadIdx.y;

    int global_idx = blockIdx.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;
    int i = global_idx / cols;
    int j = global_idx % cols;

    if (i >= rows || j >= cols) return;

    // result shape is [rows*2]
    if (d_tensor[i * cols + j] == d_int) {
        result[2*i] = i;
        result[2*i + 1] = j;
    }
}

at::Tensor findIndex(at::Tensor d_tensor, int rows, int cols, int d_int)
{
    int *count;
    hipMallocManaged(&count, sizeof(int));
    // hipDeviceSynchronize(); 
    hipMemset(count, 0, sizeof(int));
    // *count = 0;
    // int *count = &d_count;

    // compute the number of elements equal to d_int
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((rows + threadsPerBlock.x - 1) / threadsPerBlock.x, (cols + threadsPerBlock.y - 1) / threadsPerBlock.y);
    countIndexKernel<<<blocksPerGrid, threadsPerBlock>>>(d_tensor.data_ptr<int>(), rows, cols, d_int, count);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
    }
    // printf("count: ", *count, "\n");
    printf("count: %d", *count, "\n");

    // allocate memory for the result
    int* result;
    hipMallocManaged(&result, *count * 2 * sizeof(int));
    // printf("count: ", *count, "\n");

    // find the index of the elements equal to d_int
    findIndexKernel<<<blocksPerGrid, threadsPerBlock>>>(d_tensor.data_ptr<int>(), rows, cols, d_int, result);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
    }

    torch::Tensor result_tensor = torch::from_blob(result, {*count, 2}, torch::kInt32);

    hipFree(result);
    hipFree(count);

    return result_tensor;

}

__global__ void find_coords(
    const int * __restrict__ segment,
    const int * __restrict__ segment_unique,
    const int * __restrict__ segment_count,
    const int * __restrict__ pos,
    int H,
    int W,
    int num_segment,
    int* results
){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_segment) return;

    int count = segment_count[idx];
    int segment_idx = segment_unique[idx];
    int start = pos[idx];
    __syncthreads();

    int num_store = 0;
    #pragma unroll
    for (int i=0; i<H; ++i)
    {
        #pragma unroll
        for (int j=0; j<W; ++j)
        {
            if (segment[i*W+j] == segment_idx)
            {
                results[start*2 + num_store*2] = i;
                results[start*2 + num_store*2 + 1] = j;
                num_store += 1;
            }
        }
    }
}

torch::Tensor generate_sample(torch::Tensor segment,
                    torch::Tensor segment_unique,
                    torch::Tensor segment_count,
                    int H,
                    int W,
                    int num_sample,
                    float sample_ratio){
    int num_segment = segment_unique.size(0);
    int num_point = segment.size(0);

    int sample_count = 0;
    thrust::device_vector<int> segment_count_thrust(segment_count.data_ptr<int>(), segment_count.data_ptr<int>() + num_segment);
    sample_count = thrust::reduce(segment_count_thrust.begin(), segment_count_thrust.end());
    // for (int i=0; i<num_segment; ++i)
    // {
    //     int count = segment_count[i].item<int>();
    //     // sample_count += int(count * sample_ratio);
    //     sample_count += count;
    // }

    int* pixel_coords;
    hipMallocManaged(&pixel_coords, sample_count * 2 * sizeof(int));

    int* pos;
    hipMallocManaged(&pos, num_segment * sizeof(int));

    // thrust::device_vector<int> segment_count_thrust(segment_count.data_ptr<int>(), segment_count.data_ptr<int>() + num_segment);
    // thrust::devi segment_count_thrust(segment_count.data_ptr<int>(), segment_count.data_ptr<int>() + num_segment);
    for (int i=0; i<num_segment; ++i)
    {
        pos[i] = thrust::reduce(segment_count_thrust.begin(), segment_count_thrust.begin() + i);
    }

    dim3 threadsPerBlock(32, 32);
    dim3 blocksPerGrid((H + threadsPerBlock.x - 1) / threadsPerBlock.x, (W + threadsPerBlock.y - 1) / threadsPerBlock.y);
    find_coords<<<blocksPerGrid, threadsPerBlock>>>(segment.data_ptr<int>(), 
                                                    segment_unique.data_ptr<int>(), 
                                                    segment_count.data_ptr<int>(), 
                                                    pos,
                                                    H, W, num_segment, pixel_coords);
    hipDeviceSynchronize();
    
    torch::Tensor pixel_torch = torch::from_blob(pixel_coords, {sample_count, 2}, torch::kInt32).clone();

    hipFree(pixel_coords);
    hipFree(pos);

    return pixel_torch;

}

// __global__ void generate_uniform_sample(
//     torch::Tensor segments,
//     torch::Tensor segments_unique,
//     int num_segment,
//     int num_sample,
//     torch::Tensor result_coords,
//     torch::Tensor result_segments)
// {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     if (idx >= num_segment) return;

//     int segment_idx = segments_unique[idx].item<int>();
    

// }

void check_random_init(int min, int max, int N, int* results)
{
    int* d_results;
    hipMalloc((void**)&d_results, N * sizeof(int));

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    generate_random_int<<<blocksPerGrid, threadsPerBlock>>>(min, max, N, d_results);

    hipMemcpy(results, d_results, N * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_results);
}